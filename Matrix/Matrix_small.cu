#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h> 
#include <stdio.h>
#include<iostream>
#include<cstdlib>
#include<time.h> 
#include <math.h>
#define Row  150
#define Col 150

 
__global__ void matrix_mul_gpu(int *M1, int* N1, int *M2, int* N2, int* P1,int* P2 ,int width)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int aaa=213;
    int bbb=111;
    int *m=&aaa;
    int *n=&bbb;
                
    int sum = 0;
    for(int k=0;k<width;k++)
    {
        int a = M1[j*width+k];
        int b = N1[k*width+i];
        sum += a*b;
    }
    for(int k=0;k<width;k++)
    {
        int a = M2[j*width+k];
        int b = N2[k*width+i];
        sum += a*b;
    }
    /*for(int k=0;k<width;k++)
    {
        int a = M1[j*width+k];
        int b = N1[k*width+i];
        sum += a*b;
    }
    for(int k=0;k<width;k++)
    {
        int a = M2[j*width+k];
        int b = N2[k*width+i];
        sum += a*b;
    }*/
    P1[j*width+i] = sum;
   /* for(int k=0;k<width;k++)
    {
        int a = M1[j*width+k];
        int b = N2[k*width+i];
        sum += a*b;
    }
    for(int k=0;k<width;k++)
    {
        int a = M2[j*width+k];
        int b = N1[k*width+i];
        sum += a*b;
    }
    for(int k=0;k<width;k++)
    {
        int a = M1[j*width+k];
        int b = N2[k*width+i];
        sum += a*b;
    }
    for(int k=0;k<width;k++)
    {
        int a = M2[j*width+k];
        int b = N1[k*width+i];
        sum += a*b;
    }
    P2[j*width+i] = sum;*/
    asm("addc.s32 %0, %1, %2;" : "=r"(*m) : "r"(*m) , "r"(*n));
}
 
int main()
{
    struct timeval start, end;
    gettimeofday( &start, NULL );

    int *A1 = (int *)malloc(sizeof(int) * Row * Col);
    int *B1 = (int *)malloc(sizeof(int) * Row * Col);
    int *A2 = (int *)malloc(sizeof(int) * Row * Col);
    int *B2 = (int *)malloc(sizeof(int) * Row * Col);

    int *B3 = (int *)malloc(sizeof(int) * Row * Col);
    int *B4 = (int *)malloc(sizeof(int) * Row * Col);
    int *C = (int *)malloc(sizeof(int) * Row * Col);
    int *D = (int *)malloc(sizeof(int) * Row * Col);
    //malloc device memory
    int *d_dataA1, *d_dataB1,*d_dataA2, *d_dataB2,*d_dataB3, *d_dataB4, *d_dataC,*d_dataD;
    hipMalloc((void**)&d_dataA1, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataB1, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataA2, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataB2, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataB3, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataB4, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataC, sizeof(int) *Row*Col);
    hipMalloc((void**)&d_dataD, sizeof(int) *Row*Col);
    //set value
    for (int i = 0; i < Row*Col; i++) {
	srand(time(0));
        A1[i] = rand()%10000;
	srand(time(0));
        B1[i] = rand()%10000;
	srand(time(0));
        A2[i] = rand()%10000;
	srand(time(0));
        B2[i] = rand()%10000;
	srand(time(0));
	B3[i] = rand()%10000;
	srand(time(0));
	B4[i] =  rand()%10000;
    }
                                                                
    hipMemcpy(d_dataA1, A2, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB1, B1, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataA2, A2, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB2, B2, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB3, B3, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    hipMemcpy(d_dataB4, B4, sizeof(int) * Row * Col, hipMemcpyHostToDevice);
    dim3 threadPerBlock(16,16);
    dim3 blockNumber((Col+threadPerBlock.x-1)/ threadPerBlock.x, (Row+threadPerBlock.y-1)/threadPerBlock.y );
    printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA1, d_dataB1,d_dataA2, d_dataB2, d_dataC,d_dataD, Col);
    matrix_mul_gpu << <blockNumber, threadPerBlock >> > (d_dataA1, d_dataB3,d_dataA2, d_dataB4, d_dataC,d_dataD, Col);
//拷贝计算数据-一级数据指针
    hipMemcpy(C, d_dataC, sizeof(int) * Row * Col, hipMemcpyDeviceToHost);
                                                                                             
    //释放内存
    free(A1);
    free(B1);
    free(A2);
    free(B2);
    free(C);
    free(D);
    hipFree(d_dataA1);
    hipFree(d_dataB1);
    hipFree(d_dataA2);
    hipFree(d_dataB2);
    hipFree(d_dataC);
    hipFree(d_dataD);
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    printf("total time is %d ms\n", timeuse/1000);

    return 0;
}
